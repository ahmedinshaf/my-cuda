#include <stdio.h>
#include <hip/hip_runtime.h>

_global_ void vecAdd( float *A, float *B,float *C) {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        C[i] = A[i] + B[i];
}

int main(int argc,char** argv){

        int N=atoi(argv[1]); //no. of total elements
        int thread= atoi(argv[2]);
        float *a, *b, *c;
        float *dev_a, *dev_b, *dev_c;

        a = (float*)malloc(sizeof(float) * N);
        b = (float*)malloc(sizeof(float) * N);
        c = (float*)malloc(sizeof(float) * N);

        for(int i = 0 ; i < N; i++){
                a[i] = i*1.0+1;
                b[i] = i*1.0+1;
        }

        hipMalloc((void**)&dev_a, sizeof(float) * N);
        hipMalloc((void**)&dev_b, sizeof(float) * N);
        hipMalloc((void**)&dev_c, sizeof(float) * N);

        hipMemcpy(dev_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
        hipMemcpy(dev_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

        int block = (N + thread-1) / thread;
        vecAdd<<<block,thread>>>(dev_a,dev_b,dev_c);

        hipMemcpy(c, dev_c, sizeof(float) * N, hipMemcpyDeviceToHost);

        printf("Thread Size = %d\n",thread);
        printf("Block Size = %d\n",block);

        //for(int i = 0; i <N ; i++){
        //      printf("%f + %f = %f\n",a[i],b[i],c[i]);
        //}
        for(int i =0 ; i <5 ; i++){
                printf("%f + %f = %f\n",a[i],b[i],c[i]);
  }

        hipFree(dev_a);
        hipFree(dev_b);
        hipFree(dev_c);

        return 0;
}